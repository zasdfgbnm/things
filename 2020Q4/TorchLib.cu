
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>


enum class TypeKind {
  BoolType
};

struct Type;
using TypePtr = std::shared_ptr<Type>;

struct Type : std::enable_shared_from_this<Type> {
private:
 TypeKind kind_;

protected:
 Type(TypeKind kind) : kind_(kind) {}
};


struct BoolType;
using BoolTypePtr = std::shared_ptr<BoolType>;
// This node represents a Python bool value
struct BoolType : public Type {
  static const TypeKind Kind = TypeKind::BoolType;
  // global singleton
  static BoolTypePtr get() {
    return BoolTypePtr(new BoolType());
  }

 private:
  BoolType() : Type(TypeKind::BoolType) {}
};

template <typename T>
struct getTypePtr_ final {
  static TypePtr call() {
    return BoolType::get();
  }
};

struct ArgumentDef final {
  using GetTypeFn = TypePtr();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    // Create the return value
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<std::decay_t<Ts>>::call}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
