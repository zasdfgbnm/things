
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>

struct BoolType {};

template <typename T>
std::shared_ptr<BoolType> getTypePtr_() {
  return nullptr;
}

struct ArgumentDef final {
  using GetTypeFn = std::shared_ptr<BoolType>();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<Ts>}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
