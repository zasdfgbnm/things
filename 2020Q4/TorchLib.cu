
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>

template <typename T>
std::shared_ptr<bool> getTypePtr_() {
  return nullptr;
}

struct ArgumentDef final {
  using GetTypeFn = std::shared_ptr<bool>();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<Ts>}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
