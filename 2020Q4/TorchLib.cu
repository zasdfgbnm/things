
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>

struct Type;
using TypePtr = std::shared_ptr<Type>;

struct Type : std::enable_shared_from_this<Type> {
};


struct BoolType;
using BoolTypePtr = std::shared_ptr<BoolType>;
struct BoolType : public Type {
  static BoolTypePtr get() {
    return BoolTypePtr(new BoolType());
  }
};

template <typename T>
struct getTypePtr_ final {
  static TypePtr call() {
    return BoolType::get();
  }
};

struct ArgumentDef final {
  using GetTypeFn = TypePtr();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    // Create the return value
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<std::decay_t<Ts>>::call}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
