
#include <hip/hip_runtime.h>
#include <utility>
#include <array>

template <typename T>
bool getTypePtr_() {
  return false;
}

struct ArgumentDef final {
  using GetTypeFn = bool();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, std::size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<Ts>}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
