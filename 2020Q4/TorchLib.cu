
#include <hip/hip_runtime.h>
#include <utility>
#include <array>

struct ArgumentDef final {
  std::size_t i;
};

template <std::size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Is)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    std::array<ArgumentDef, sizeof...(Is)>{{ArgumentDef{Is}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes(std::make_index_sequence<1>());
}
