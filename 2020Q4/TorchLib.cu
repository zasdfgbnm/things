
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>


enum class TypeKind {
  BoolType
};

struct Type;
using TypePtr = std::shared_ptr<Type>;
using ConstTypePtr = std::shared_ptr<const Type>;

struct Type : std::enable_shared_from_this<Type> {
private:
 TypeKind kind_;

protected:
 Type(TypeKind kind) : kind_(kind) {}

public:

 TypeKind kind() const {
   return kind_;
 }

 virtual ~Type() = default;
};


struct BoolType;
using BoolTypePtr = std::shared_ptr<BoolType>;
// This node represents a Python bool value
struct BoolType : public Type {
  static const TypeKind Kind = TypeKind::BoolType;
  // global singleton
  static BoolTypePtr get() {
    return BoolTypePtr(new BoolType());
  }

 private:
  BoolType() : Type(TypeKind::BoolType) {}
};

template <typename T>
struct getTypePtr_ final {
  static TypePtr call() {
    return BoolType::get();
  }
};

struct ArgumentDef final {
  using GetTypeFn = TypePtr();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    // Create the return value
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<std::decay_t<Ts>>::call}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
