
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>

struct BoolType {};

template <typename T>
struct getTypePtr_ final {
  static std::shared_ptr<BoolType> call() {
    return nullptr;
  }
};

struct ArgumentDef final {
  using GetTypeFn = std::shared_ptr<BoolType>();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    // Create the return value
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<Ts>::call}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
