
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <type_traits>
#include <array>

struct BoolType;
using BoolTypePtr = std::shared_ptr<BoolType>;
struct BoolType {
  static BoolTypePtr get() {
    return BoolTypePtr(new BoolType());
  }
};

template <typename T>
struct getTypePtr_ final {
  static BoolTypePtr call() {
    return BoolType::get();
  }
};

struct ArgumentDef final {
  using GetTypeFn = BoolTypePtr();
  GetTypeFn* getTypeFn;
};

template <typename... Ts, size_t... Is>
constexpr std::array<ArgumentDef, sizeof...(Ts)> createArgumentVectorFromTypes(std::index_sequence<Is...>) {
  return (
    // Create the return value
    std::array<ArgumentDef, sizeof...(Ts)>{{ArgumentDef{&getTypePtr_<Ts>::call}...}}
  );
}

int main() {
    constexpr auto returns = createArgumentVectorFromTypes<bool>(std::make_index_sequence<1>());
}
