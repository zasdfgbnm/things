#include "hip/hip_runtime.h"
#include <cassert>
#include <cudnn_frontend.h>
#include <random>
#include <vector>
#include <iostream>

constexpr int64_t DIMS = 4;

struct Tensor {
  int64_t shape[DIMS];
  int64_t strides[DIMS];
  float *data;
  float &operator()(int64_t i, int64_t j, int64_t k, int64_t l) {
    int64_t index =
        i * strides[0] + j * strides[1] + k * strides[2] + l * strides[3];
    return data[index];
  }
};

std::ostream &operator<<(std::ostream &out, const Tensor &t) {
    return (out << "Tensor(shape=[" << t.shape[0] << "," << t.shape[1] << "," << t.shape[2] << "," << t.shape[3] \
        << "], stride=[" << t.strides[0] << "," << t.strides[1] << "," << t.strides[2] << "," << t.strides[3] \
        << "])");
}

Tensor new_tensor(const std::vector<int64_t> &shape,
                  const std::vector<int64_t> &dim_order) {
  Tensor ret;

  for (int i = 0; i < DIMS; i++) {
    ret.shape[i] = shape[i];
  }

  int64_t size = 1;
  for (int i = 0; i < DIMS; i++) {
    auto dim = dim_order[i];
    ret.strides[dim] = size;
    size *= shape[dim];
  }
  hipMallocManaged(&ret.data, size * sizeof(float));
  return ret;
}

void copy(Tensor &to, Tensor &from) {
  for (int i = 0; i < from.shape[0]; i++) {
    for (int j = 0; j < from.shape[1]; j++) {
      for (int k = 0; k < from.shape[2]; k++) {
        for (int l = 0; l < from.shape[3]; l++) {
          to(i, j, k, l) = from(i, j, k, l);
        }
      }
    }
  }
}

float maxdiff(Tensor &to, Tensor &from) {
  float result = -1;
  for (int i = 0; i < from.shape[0]; i++) {
    for (int j = 0; j < from.shape[1]; j++) {
      for (int k = 0; k < from.shape[2]; k++) {
        for (int l = 0; l < from.shape[3]; l++) {
          float diff = std::abs(to(i, j, k, l) - from(i, j, k, l));
          if (diff > result) {
            result = diff;
          }
        }
      }
    }
  }
  return result;
}

void random_fill(Tensor &t) {
  std::default_random_engine generator;
  std::uniform_real_distribution<float> distribution(0.0, 1.0);
  for (int i = 0; i < t.shape[0]; i++) {
    for (int j = 0; j < t.shape[1]; j++) {
      for (int k = 0; k < t.shape[2]; k++) {
        for (int l = 0; l < t.shape[3]; l++) {
          t(i, j, k, l) = distribution(generator);
        }
      }
    }
  }
}

class CuDNNError : public std::runtime_error {
    using runtime_error::runtime_error;
};

#define CUDNN_CHECK(EXPR, ...)                                                 \
  do {                                                                         \
    hipdnnStatus_t status = EXPR;                                               \
    if (status != HIPDNN_STATUS_SUCCESS) {                                      \
      throw CuDNNError("cuDNN error");                                         \
    }                                                                          \
  } while (0)

uint8_t getAlignment(const Tensor &t) {
  // alignment are in bytes
  uint8_t alignment = 1;
  uint64_t address = reinterpret_cast<uint64_t>(t.data);
  while (address % alignment == 0 && alignment < 16)
    alignment *= 2;
  return alignment;
}

cudnn_frontend::Tensor getTensorDescriptor(const Tensor &t, int64_t id) {
  return cudnn_frontend::TensorBuilder()
      .setDim(DIMS, t.shape)
      .setStrides(DIMS, t.strides)
      .setId(id)
      .setAlignment(getAlignment(t))
      .setDataType(HIPDNN_DATA_FLOAT)
      .build();
}

cudnn_frontend::ConvDesc_v8 getConvDescriptor(std::vector<int64_t> padding,
                                              std::vector<int64_t> stride,
                                              std::vector<int64_t> dilation) {
  uint64_t convDim = stride.size();
  return cudnn_frontend::ConvDescBuilder()
      .setDataType(HIPDNN_DATA_FLOAT)
      .setMathMode(HIPDNN_CROSS_CORRELATION)
      .setNDims(convDim)
      .setStrides(convDim, stride.data())
      .setPrePadding(convDim, padding.data())
      .setPostPadding(convDim, padding.data())
      .setDilation(convDim, dilation.data())
      .build();
}

void filterEngineConfigs(cudnn_frontend::EngineConfigList &from,
                         cudnn_frontend::EngineConfigList &to,
                         bool deterministic, bool allow_tf32) {
  auto filter = [=](cudnnBackendDescriptor_t c) {
    if (deterministic) {
      if (cudnn_frontend::hasNumericalNote<
              CUDNN_NUMERICAL_NOTE_NONDETERMINISTIC>(c))
        return true;
    }
    if (!allow_tf32) {
      if (cudnn_frontend::hasNumericalNote<
              CUDNN_NUMERICAL_NOTE_DOWN_CONVERT_INPUTS>(c))
        return true;
      if (cudnn_frontend::hasNumericalNote<CUDNN_NUMERICAL_NOTE_TENSOR_CORE>(c))
        return true;
    }
    return false;
  };
  cudnn_frontend::filter(from, to, filter);
}

void convolution(Tensor input, Tensor weight, Tensor output,
                 std::vector<int64_t> padding, std::vector<int64_t> stride,
                 std::vector<int64_t> dilation, bool deterministic,
                 bool allow_tf32) {
  hipdnnHandle_t handle;
  hipdnnCreate(&handle);

  auto run = [&](cudnn_frontend::ManagedOpaqueDescriptor cfg) {
    auto plan = cudnn_frontend::ExecutionPlanBuilder()
                    .setHandle(handle)
                    .setEngineConfig(cfg)
                    .build();

    auto workspace_size = plan.getWorkspaceSize();
    void *workspace;
    hipMalloc(&workspace, workspace_size);
    void *data_ptrs[] = {input.data, output.data, weight.data};

    int64_t uids[] = {'x', 'y', 'w'};
    auto variantPack = cudnn_frontend::VariantPackBuilder()
                           .setWorkspacePointer(workspace)
                           .setDataPointers(3, data_ptrs)
                           .setUids(3, uids)
                           .build();
    CUDNN_CHECK(cudnnBackendExecute(handle, plan.get_raw_desc(),
                                    variantPack.get_raw_desc()));
  };

  auto op = cudnn_frontend::OperationBuilder(
                CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
                .setxDesc(getTensorDescriptor(input, 'x'))
                .setyDesc(getTensorDescriptor(output, 'y'))
                .setwDesc(getTensorDescriptor(weight, 'w'))
                .setcDesc(getConvDescriptor(padding, stride, dilation))
                .build();
  // std::cout << op.describe() << std::endl;

  std::array<cudnn_frontend::Operation const *, 1> ops = {&op};

  auto opGraph = cudnn_frontend::OperationGraphBuilder()
                     .setHandle(handle)
                     .setOperationGraph(1, ops.data())
                     .build();
  // std::cout << opGraph.describe() << std::endl;

  auto heuristics = cudnn_frontend::EngineHeuristicsBuilder()
                        .setOperationGraph(opGraph)
                        .setHeurMode(CUDNN_HEUR_MODE_INSTANT)
                        .build();
  auto fallback =
      cudnn_frontend::EngineFallbackListBuilder()
          .setOperationGraph(opGraph)
          .setOperation(CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
          .build();

  auto &engine_configs =
      heuristics.getEngineConfig(heuristics.getEngineConfigCount());
  auto &fallback_list = fallback.getFallbackList();

  cudnn_frontend::EngineConfigList filtered_configs;
  filterEngineConfigs(engine_configs, filtered_configs, deterministic,
                      allow_tf32);
  filterEngineConfigs(fallback_list, filtered_configs, deterministic,
                      allow_tf32);

  for (auto &cfg : filtered_configs) {
    try {
      run(cfg);
      return;
    } catch (cudnn_frontend::cudnnException &e) {
    } catch (CuDNNError &e) {
    }
  }
}

int main() {
  Tensor input = new_tensor({2, 8, 4, 4}, {3, 2, 1, 0});
  random_fill(input); std::cout << input << std::endl;
  Tensor weight = new_tensor({4, 8, 3, 3}, {3, 2, 1, 0});
  random_fill(weight); std::cout << weight << std::endl;
  Tensor output = new_tensor({2, 4, 2, 2}, {3, 2, 1, 0});
  std::cout << output << std::endl;
  std::vector<int64_t> padding = {0, 0};
  std::vector<int64_t> stride = {1, 1};
  std::vector<int64_t> dilation = {1, 1};
  convolution(input, weight, output, padding, stride, dilation, false, true);

  Tensor input2 = new_tensor({2, 8, 4, 4}, {1, 3, 2, 0});
  copy(input2, input); std::cout << input << std::endl;
  Tensor weight2 = new_tensor({4, 8, 3, 3}, {1, 3, 2, 0});
  copy(weight2, weight); std::cout << weight << std::endl;
  Tensor output2 = new_tensor({2, 4, 2, 2}, {1, 3, 2, 0});
  std::cout << output << std::endl;
  convolution(input2, weight2, output2, padding, stride, dilation, false, true);

  std::cout << "diff = " << maxdiff(output, output2) << std::endl;
}
