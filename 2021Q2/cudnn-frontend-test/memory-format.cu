
#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>

constexpr int64_t DIMS = 4;

struct Tensor {
    int64_t shape[DIMS];
    int64_t strides[DIMS];
    float *data;
    float & operator()(int64_t i, int64_t j, int64_t k, int64_t l) {
        int64_t index = i * strides[0] + j * strides[1] + k * strides[2] + l * strides[3];
        return data[index];
    }
};

Tensor new_tensor(const std::vector<int64_t> &shape, const std::vector<int64_t> &dim_order) {
    Tensor ret;

    for(int i = 0; i < DIMS; i++) {
        ret.shape[i] = shape[i];
    }

    int64_t size = 1;
    for(int i = 0; i < DIMS; i++) {
        auto dim = dim_order[i];
        ret.strides[dim] = size;
        size *= shape[dim];
    }
    hipMallocManaged(&ret.data, size * sizeof(float));
    return ret;
}

void copy(Tensor &to, Tensor &from) {
    for (int i = 0; i < from.shape[0]; i++) {
        for (int j = 0; j < from.shape[1]; j++) {
            for (int k = 0; k < from.shape[2]; k++) {
                for (int l = 0; l < from.shape[3]; l++) {
                    to(i, j, k, l) = from(i, j, k, l);
                }
            }
        }
    }
}

int main() {}