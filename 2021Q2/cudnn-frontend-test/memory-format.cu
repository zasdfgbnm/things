#include "hip/hip_runtime.h"
#include <cassert>
#include <cudnn_frontend.h>
#include <random>
#include <vector>

constexpr int64_t DIMS = 4;

struct Tensor {
  int64_t shape[DIMS];
  int64_t strides[DIMS];
  float *data;
  float &operator()(int64_t i, int64_t j, int64_t k, int64_t l) {
    int64_t index =
        i * strides[0] + j * strides[1] + k * strides[2] + l * strides[3];
    return data[index];
  }
};

Tensor new_tensor(const std::vector<int64_t> &shape,
                  const std::vector<int64_t> &dim_order) {
  Tensor ret;

  for (int i = 0; i < DIMS; i++) {
    ret.shape[i] = shape[i];
  }

  int64_t size = 1;
  for (int i = 0; i < DIMS; i++) {
    auto dim = dim_order[i];
    ret.strides[dim] = size;
    size *= shape[dim];
  }
  hipMallocManaged(&ret.data, size * sizeof(float));
  return ret;
}

void copy(Tensor &to, Tensor &from) {
  for (int i = 0; i < from.shape[0]; i++) {
    for (int j = 0; j < from.shape[1]; j++) {
      for (int k = 0; k < from.shape[2]; k++) {
        for (int l = 0; l < from.shape[3]; l++) {
          to(i, j, k, l) = from(i, j, k, l);
        }
      }
    }
  }
}

float maxdiff(Tensor &to, Tensor &from) {
  float result = -1;
  for (int i = 0; i < from.shape[0]; i++) {
    for (int j = 0; j < from.shape[1]; j++) {
      for (int k = 0; k < from.shape[2]; k++) {
        for (int l = 0; l < from.shape[3]; l++) {
          float diff = std::abs(to(i, j, k, l) - from(i, j, k, l));
          if (diff > result) {
            result = diff;
          }
        }
      }
    }
  }
  return result;
}

void random_fill(Tensor &t) {
  std::default_random_engine generator;
  std::uniform_real_distribution<float> distribution(0.0, 1.0);
  for (int i = 0; i < t.shape[0]; i++) {
    for (int j = 0; j < t.shape[1]; j++) {
      for (int k = 0; k < t.shape[2]; k++) {
        for (int l = 0; l < t.shape[3]; l++) {
          t(i, j, k, l) = distribution(generator);
        }
      }
    }
  }
}

uint8_t getAlignment(const Tensor &t) {
  // alignment are in bytes
  uint8_t alignment = 1;
  uint64_t address = reinterpret_cast<uint64_t>(t.data);
  while (address % alignment == 0 && alignment < 16)
    alignment *= 2;
  return alignment;
}

cudnn_frontend::Tensor getTensorDescriptor(const Tensor &t, int64_t id) {
  return cudnn_frontend::TensorBuilder()
      .setDim(DIMS, t.shape)
      .setStrides(DIMS, t.strides)
      .setId(id)
      .setAlignment(getAlignment(t))
      .setDataType(HIPDNN_DATA_FLOAT)
      .build();
}

cudnn_frontend::ConvDesc_v8 getConvDescriptor(std::vector<int64_t> padding,
                                              std::vector<int64_t> stride,
                                              std::vector<int64_t> dilation) {
  uint64_t convDim = stride.size();
  return cudnn_frontend::ConvDescBuilder()
      .setDataType(HIPDNN_DATA_FLOAT)
      .setMathMode(HIPDNN_CROSS_CORRELATION)
      .setNDims(convDim)
      .setStrides(convDim, stride.data())
      .setPrePadding(convDim, padding.data())
      .setPostPadding(convDim, padding.data())
      .setDilation(convDim, dilation.data())
      .build();
}

void filterEngineConfigs(cudnn_frontend::EngineConfigList &from,
                         cudnn_frontend::EngineConfigList &to,
                         bool deterministic, bool allow_tf32) {
  auto filter = [=](cudnnBackendDescriptor_t c) {
    if (deterministic) {
      if (cudnn_frontend::hasNumericalNote<
              CUDNN_NUMERICAL_NOTE_NONDETERMINISTIC>(c))
        return true;
    }
    if (!allow_tf32) {
      if (cudnn_frontend::hasNumericalNote<
              CUDNN_NUMERICAL_NOTE_DOWN_CONVERT_INPUTS>(c))
        return true;
      if (cudnn_frontend::hasNumericalNote<CUDNN_NUMERICAL_NOTE_TENSOR_CORE>(c))
        return true;
    }
    return false;
  };
  cudnn_frontend::filter(from, to, filter);
}

void convolution(Tensor input, Tensor weight, Tensor output,
                 std::vector<int64_t> padding, std::vector<int64_t> stride,
                 std::vector<int64_t> dilation, bool deterministic,
                 bool allow_tf32) {
  hipdnnHandle_t handle;
  hipdnnCreate(&handle);

  auto run = [&](cudnn_frontend::ManagedOpaqueDescriptor cfg) {
    auto plan = cudnn_frontend::ExecutionPlanBuilder()
                    .setHandle(handle)
                    .setEngineConfig(cfg)
                    .build();

    auto workspace_size = plan.getWorkspaceSize();
    void *workspace;
    hipMalloc(&workspace, workspace_size);
    void *data_ptrs[] = {input.data, output.data, weight.data};

    int64_t uids[] = {'x', 'y', 'w'};
    auto variantPack = cudnn_frontend::VariantPackBuilder()
                           .setWorkspacePointer(workspace)
                           .setDataPointers(3, data_ptrs)
                           .setUids(3, uids)
                           .build();
    cudnnBackendExecute(handle, plan.get_raw_desc(),
                        variantPack.get_raw_desc());
  };

  auto op = cudnn_frontend::OperationBuilder(
                CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
                .setxDesc(getTensorDescriptor(input, 'x'))
                .setyDesc(getTensorDescriptor(output, 'y'))
                .setwDesc(getTensorDescriptor(weight, 'w'))
                .setcDesc(getConvDescriptor(padding, stride, dilation))
                .build();
  // std::cout << op.describe() << std::endl;

  std::array<cudnn_frontend::Operation const *, 1> ops = {&op};

  auto opGraph = cudnn_frontend::OperationGraphBuilder()
                     .setHandle(handle)
                     .setOperationGraph(1, ops.data())
                     .build();
  // std::cout << opGraph.describe() << std::endl;

  auto heuristics = cudnn_frontend::EngineHeuristicsBuilder()
                        .setOperationGraph(opGraph)
                        .setHeurMode(CUDNN_HEUR_MODE_INSTANT)
                        .build();
  auto fallback =
      cudnn_frontend::EngineFallbackListBuilder()
          .setOperationGraph(opGraph)
          .setOperation(CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
          .build();

  auto &engine_configs =
      heuristics.getEngineConfig(heuristics.getEngineConfigCount());
  auto &fallback_list = fallback.getFallbackList();

  cudnn_frontend::EngineConfigList filtered_configs;
  filterEngineConfigs(engine_configs, filtered_configs, deterministic,
                      allow_tf32);
  filterEngineConfigs(fallback_list, filtered_configs, deterministic,
                      allow_tf32);

  for (auto &cfg : filtered_configs) {
    try {
      run(cfg);
      return;
    } catch (cudnn_frontend::cudnnException &e) {
    }
  }
}

int main() {}
