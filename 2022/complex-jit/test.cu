
#include <hip/hip_runtime.h>
#include "complex"

__global__ void kernel(std::complex<float> *in, std::complex<float> *out) {
    *out = std::sin(*in);
}

int main() {
    kernel<<<1,1>>>(nullptr, nullptr);
}
