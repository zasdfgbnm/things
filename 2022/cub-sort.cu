#include <chrono>
#include <hipcub/hipcub.hpp>

template <int N> struct alignas(N) OpaqueType { char data[N]; };

using KeyT = float;
using ValueT = OpaqueType<8>;

void init(size_t N, KeyT *h_keys, ValueT *h_values) {
  for (size_t i = 0; i < N; i++) {
    h_keys[i] = (i % 10);
    for (auto &c : h_values[i].data) {
      c = (char)i;
    }
  }
}

void run(size_t N) {
  KeyT *h_keys;
  ValueT *h_values;
  h_keys = new KeyT[N];
  h_values = new ValueT[N];
  init(N, h_keys, h_values);

  KeyT *d_keys;
  hipMalloc(&d_keys, sizeof(KeyT) * N);
  hipMemcpy(d_keys, h_keys, sizeof(KeyT) * N, hipMemcpyDefault);

  ValueT *d_values;
  hipMalloc(&d_values, sizeof(ValueT) * N);
  hipMemcpy(d_values, h_values, sizeof(ValueT) * N, hipMemcpyDefault);

  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, d_keys, d_keys,
                                  d_values, d_values, N);

  void *d_temp_storage;
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point begin =
      std::chrono::steady_clock::now();
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys,
                                  d_keys, d_values, d_values, N);
  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout << "Problem size = " << N << std::endl
            << "Time difference = "
            << std::chrono::duration_cast<std::chrono::microseconds>(end -
                                                                     begin)
                   .count()
            << "[µs]" << std::endl;
  hipFree(d_keys);
  hipFree(d_values);
  hipFree(d_temp_storage);
  delete[] h_keys;
  delete[] h_values;
}

int main() {
  for (size_t N = 10; N <= 100'000'000; N *= 10) {
    run(N);
  }
}

// nsys nvprof nvcc -std=c++14 -gencode arch=compute_70,code=sm_70 -gencode
// arch=compute_80,code=sm_80 -run cub-sort.cu

// nvcc -std=c++14 -gencode arch=compute_70,code=sm_70 -gencode
// arch=compute_80,code=sm_80 -run cub-sort.cu
