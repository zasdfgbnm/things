#include <hipcub/hipcub.hpp>
#include <chrono>

template <int N> struct alignas(N) OpaqueType { char data[N]; };

using KeyT = float;
using ValueT = OpaqueType<8>;
constexpr size_t N = 100'000'000;

KeyT h_keys[N];
ValueT h_values[N];

void init() {
  for (size_t i = 0; i < N; i++) {
    h_keys[i] = (i % 10);
    for (auto &c : h_values[i].data) {
      c = (char)i;
    }
  }
}

int main() {
  KeyT *d_keys;
  hipMalloc(&d_keys, sizeof(KeyT) * N);
  hipMemcpy(d_keys, h_keys, sizeof(KeyT) * N, hipMemcpyDefault);

  ValueT *d_values;
  hipMalloc(&d_values, sizeof(ValueT) * N);
  hipMemcpy(d_values, h_values, sizeof(ValueT) * N, hipMemcpyDefault);

  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, d_keys, d_keys, d_values, d_values, N);

  void *d_temp_storage;
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys, d_values, d_values, N);
  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
}

// nsys nvprof nvcc -std=c++14 -arch=sm_80 -run cub-sort.cu
// nvcc -std=c++14 -arch=sm_80 -run cub-sort.cu