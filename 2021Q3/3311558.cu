#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <cudnn_frontend.h>
#include <iostream>
#include <random>
#include <vector>

constexpr int64_t DIMS = 4;

template <typename T> struct Tensor {
  using dtype = T;
  int64_t shape[DIMS];
  int64_t strides[DIMS];
  T *data;
  T &operator()(int64_t i, int64_t j, int64_t k, int64_t l) {
    int64_t index =
        i * strides[0] + j * strides[1] + k * strides[2] + l * strides[3];
    return data[index];
  }
  const T &operator()(int64_t i, int64_t j, int64_t k, int64_t l) const {
    int64_t index =
        i * strides[0] + j * strides[1] + k * strides[2] + l * strides[3];
    return data[index];
  }
};

template <typename T>
std::ostream &operator<<(std::ostream &out, const Tensor<T> &t) {
  out << "Tensor(shape=[" << t.shape[0] << "," << t.shape[1] << ","
      << t.shape[2] << "," << t.shape[3] << "], stride=[" << t.strides[0] << ","
      << t.strides[1] << "," << t.strides[2] << "," << t.strides[3]
      << "], data = [\n";
  bool firsti = true;
  for (int64_t i = 0; i < t.shape[0]; i++) {
    if (!firsti) {
      out << ",\n";
    }
    out << " [";
    bool firstj = true;
    for (int64_t j = 0; j < t.shape[1]; j++) {
      if (!firstj) {
        out << ",\n  ";
      }
      out << "[";
      bool firstk = true;
      for (int64_t k = 0; k < t.shape[2]; k++) {
        if (!firstk) {
          out << ", ";
        }
        out << "[";
        bool firstl = true;
        for (int64_t l = 0; l < t.shape[3]; l++) {
          if (!firstl) {
            out << ", ";
          }
          out << t(i, j, k, l);
          firstl = false;
        }
        out << "]";
        firstk = false;
      }
      out << "]";
      firstj = false;
    }
    out << "]";
    firsti = false;
  }
  out << "]);";
  return out;
}

template <typename T> inline hipdnnDataType_t getDataType() {
  if (std::is_same<T, float>::value) {
    return HIPDNN_DATA_FLOAT;
  } else if (std::is_same<T, __half>::value) {
    return HIPDNN_DATA_HALF;
  } else if (std::is_same<T, double>::value) {
    return HIPDNN_DATA_DOUBLE;
  } else if (std::is_same<T, __hip_bfloat16>::value) {
    return CUDNN_DATA_BFLOAT16;
  }
  throw std::runtime_error(
      "TensorDescriptor only supports double, float and half tensors");
}

template <typename T>
Tensor<T> new_tensor(const std::vector<int64_t> &shape,
                     const std::vector<int64_t> &dim_order) {
  Tensor<T> ret;

  for (int i = 0; i < DIMS; i++) {
    ret.shape[i] = shape[i];
  }

  int64_t size = 1;
  for (int i = 0; i < DIMS; i++) {
    auto dim = dim_order[i];
    ret.strides[dim] = size;
    size *= shape[dim];
  }
  hipMallocManaged(&ret.data, size * sizeof(T));
  return ret;
}

template <typename T1, typename T2>
void copy(Tensor<T1> &to, Tensor<T2> &from) {
  for (int i = 0; i < from.shape[0]; i++) {
    for (int j = 0; j < from.shape[1]; j++) {
      for (int k = 0; k < from.shape[2]; k++) {
        for (int l = 0; l < from.shape[3]; l++) {
          to(i, j, k, l) = from(i, j, k, l);
        }
      }
    }
  }
}

template <typename T1, typename T2>
double maxdiff(Tensor<T1> &to, Tensor<T2> &from) {
  double result = -1;
  for (int i = 0; i < from.shape[0]; i++) {
    for (int j = 0; j < from.shape[1]; j++) {
      for (int k = 0; k < from.shape[2]; k++) {
        for (int l = 0; l < from.shape[3]; l++) {
          auto diff = std::abs(to(i, j, k, l) - from(i, j, k, l));
          if (diff > result) {
            result = diff;
          }
        }
      }
    }
  }
  return result;
}

template <typename T> void random_fill(Tensor<T> &t) {
  std::default_random_engine generator;
  std::uniform_int_distribution<int> distribution(-3, 3);
  for (int i = 0; i < t.shape[0]; i++) {
    for (int j = 0; j < t.shape[1]; j++) {
      for (int k = 0; k < t.shape[2]; k++) {
        for (int l = 0; l < t.shape[3]; l++) {
          t(i, j, k, l) = distribution(generator);
        }
      }
    }
  }
}

class CuDNNError : public std::runtime_error {
  using runtime_error::runtime_error;
};

#define CUDNN_CHECK(EXPR, ...)                                                 \
  do {                                                                         \
    hipdnnStatus_t status = EXPR;                                               \
    if (status != HIPDNN_STATUS_SUCCESS) {                                      \
      throw CuDNNError("cuDNN error");                                         \
    }                                                                          \
  } while (0)

template <typename T> uint8_t getAlignment(const Tensor<T> &t) {
  // alignment are in bytes
  uint8_t alignment = 1;
  uint64_t address = reinterpret_cast<uint64_t>(t.data);
  while (address % alignment == 0 && alignment < 16)
    alignment *= 2;
  return alignment;
}

template <typename T>
cudnn_frontend::Tensor getTensorDescriptor(const Tensor<T> &t, int64_t id) {
  return cudnn_frontend::TensorBuilder()
      .setDim(DIMS, t.shape)
      .setStrides(DIMS, t.strides)
      .setId(id)
      .setAlignment(getAlignment(t))
      .setDataType(getDataType<T>())
      .build();
}

cudnn_frontend::ConvDesc_v8 getConvDescriptor(hipdnnDataType_t dtype,
                                              std::vector<int64_t> padding,
                                              std::vector<int64_t> stride,
                                              std::vector<int64_t> dilation) {
  uint64_t convDim = stride.size();
  return cudnn_frontend::ConvDescBuilder()
      .setDataType(dtype)
      .setMathMode(HIPDNN_CROSS_CORRELATION)
      .setNDims(convDim)
      .setStrides(convDim, stride.data())
      .setPrePadding(convDim, padding.data())
      .setPostPadding(convDim, padding.data())
      .setDilation(convDim, dilation.data())
      .build();
}

void filterEngineConfigs(cudnn_frontend::EngineConfigList &from,
                         cudnn_frontend::EngineConfigList &to,
                         bool deterministic, bool allow_tf32) {
  auto filter = [=](cudnnBackendDescriptor_t c) {
    if (deterministic) {
      if (cudnn_frontend::hasNumericalNote<
              CUDNN_NUMERICAL_NOTE_NONDETERMINISTIC>(c))
        return true;
    }
    if (!allow_tf32) {
      if (cudnn_frontend::hasNumericalNote<
              CUDNN_NUMERICAL_NOTE_DOWN_CONVERT_INPUTS>(c))
        return true;
      if (cudnn_frontend::hasNumericalNote<CUDNN_NUMERICAL_NOTE_TENSOR_CORE>(c))
        return true;
    }
    return false;
  };
  cudnn_frontend::filter(from, to, filter);
}

template <typename T1, typename T2, typename T3>
void convolution(Tensor<T1> input, Tensor<T2> weight, Tensor<T3> output,
                 std::vector<int64_t> padding, std::vector<int64_t> stride,
                 std::vector<int64_t> dilation, bool deterministic,
                 bool allow_tf32) {
  hipdnnHandle_t handle;
  hipdnnCreate(&handle);

  auto run = [&](cudnn_frontend::ManagedOpaqueDescriptor cfg) {
    auto plan = cudnn_frontend::ExecutionPlanBuilder()
                    .setHandle(handle)
                    .setEngineConfig(cfg)
                    .build();

    auto workspace_size = plan.getWorkspaceSize();
    void *workspace;
    hipMalloc(&workspace, workspace_size);
    void *data_ptrs[] = {input.data, output.data, weight.data};

    int64_t uids[] = {'x', 'y', 'w'};
    auto variantPack = cudnn_frontend::VariantPackBuilder()
                           .setWorkspacePointer(workspace)
                           .setDataPointers(3, data_ptrs)
                           .setUids(3, uids)
                           .build();
    CUDNN_CHECK(cudnnBackendExecute(handle, plan.get_raw_desc(),
                                    variantPack.get_raw_desc()));
  };

  auto op = cudnn_frontend::OperationBuilder(
                CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
                .setxDesc(getTensorDescriptor(input, 'x'))
                .setyDesc(getTensorDescriptor(output, 'y'))
                .setwDesc(getTensorDescriptor(weight, 'w'))
                .setcDesc(getConvDescriptor(getDataType<T3>(), padding, stride,
                                            dilation))
                .build();
  // std::cout << op.describe() << std::endl;

  std::array<cudnn_frontend::Operation const *, 1> ops = {&op};

  auto opGraph = cudnn_frontend::OperationGraphBuilder()
                     .setHandle(handle)
                     .setOperationGraph(1, ops.data())
                     .build();
  // std::cout << opGraph.describe() << std::endl;

  auto heuristics = cudnn_frontend::EngineHeuristicsBuilder()
                        .setOperationGraph(opGraph)
                        .setHeurMode(CUDNN_HEUR_MODE_INSTANT)
                        .build();
  auto fallback =
      cudnn_frontend::EngineFallbackListBuilder()
          .setOperationGraph(opGraph)
          .setOperation(CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR)
          .build();

  auto &engine_configs =
      heuristics.getEngineConfig(heuristics.getEngineConfigCount());
  auto &fallback_list = fallback.getFallbackList();

  cudnn_frontend::EngineConfigList filtered_configs;
  filterEngineConfigs(engine_configs, filtered_configs, deterministic,
                      allow_tf32);
  filterEngineConfigs(fallback_list, filtered_configs, deterministic,
                      allow_tf32);

  for (auto &cfg : filtered_configs) {
    try {
      run(cfg);
      return;
    } catch (cudnn_frontend::cudnnException &e) {
    } catch (CuDNNError &e) {
    }
  }
}

int main() {
  std::vector<int64_t> padding = {0, 0};
  std::vector<int64_t> stride = {1, 1};
  std::vector<int64_t> dilation = {1, 1};

  // float nchw
  auto input = new_tensor<float>({2, 8, 4, 4}, {3, 2, 1, 0});
  random_fill(input);
  // std::cout << "input = " << input << std::endl;
  auto weight = new_tensor<float>({4, 8, 3, 3}, {3, 2, 1, 0});
  random_fill(weight);
  auto output = new_tensor<float>({2, 4, 2, 2}, {3, 2, 1, 0});
  convolution(input, weight, output, padding, stride, dilation, false, true);

  // double nchw
  auto input2 = new_tensor<double>({2, 8, 4, 4}, {3, 2, 1, 0});
  copy(input2, input);
  auto weight2 = new_tensor<double>({4, 8, 3, 3}, {3, 2, 1, 0});
  copy(weight2, weight);
  auto output2 = new_tensor<double>({2, 4, 2, 2}, {3, 2, 1, 0});
  convolution(input2, weight2, output2, padding, stride, dilation, false, true);

  std::cout << "diff(output, output2) = " << maxdiff(output, output2)
            << std::endl;
}